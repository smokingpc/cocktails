#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <windows.h>
#include <stdio.h>
#include <stdint.h>
#include "test.cuh"

template<typename F, unsigned int shift, unsigned int count>
class CopyLoop {
	public:
	__device__ __forceinline__ static
	void _copy(F f, volatile uint32_t * const dst, volatile uint32_t * const src) {
		if(count < 512)
		{
			constexpr unsigned int_index = ((count - 1) << shift);
			f(dst + int_index, src + int_index);
			CopyLoop<F, shift, count - 1>::_copy(f, dst, src);
		}
	}
};

template<typename F, int SFT>
class CopyLoop<F, SFT, 0> {
	public:
	__device__ __forceinline__ static
	void _copy(F f, volatile uint32_t * const dst, volatile uint32_t * const src) {}
};

__device__ __forceinline__ static
void memcpy16(volatile uint32_t *d, volatile uint32_t *s)
{
	asm volatile("\n{\
		.reg .u32 d0;\n\
		.reg .u32 d1;\n\
		.reg .u32 d2;\n\
		.reg .u32 d3;\n\
		ld.cv.v4.u32 { d0,  d1,  d2,  d3}, [%0];\n\
		st.wt.v4.u32 [%1], { d0,  d1,  d2,  d3};}" \
		: : "l"(s), "l"(d) : "memory");
}


__device__ bool DoDeviceCode()
{
    constexpr uint32_t LENGTH = 16;
	constexpr uint32_t copy_nr_64bytes = LENGTH >> 6;
	constexpr uint32_t copy_nr_16bytes = (LENGTH >> 4) - (copy_nr_64bytes << 2);
    uint32_t dst16[64] = {0}, src16[64] = {0};
    CopyLoop<__typeof__(memcpy16), 2, copy_nr_16bytes>::_copy(memcpy16, dst16, src16);
    return true;
}
__global__ void DoGpuJob()
{
    DoDeviceCode();
}

__host__ void GpuThreads(void* ctx)
{
    DoGpuJob();
}

void LaunchGpuThreads(void* ctx)
{
	GpuThreads(ctx);
}
